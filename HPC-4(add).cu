// vector_add.cu

#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void vecAdd(float *A, float *B, float *C, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N)
        C[i] = A[i] + B[i];
}

int main() {
    int N = 5;
    size_t size = N * sizeof(float);
    float A[] = {1, 2, 3, 4, 5};
    float B[] = {10, 20, 30, 40, 50};
    float C[5];

    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    vecAdd<<<1, N>>>(d_A, d_B, d_C, N);

    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

    printf("Result Vector C:\n");
    for (int i = 0; i < N; i++)
        printf("%f ", C[i]);
    printf("\n");

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    return 0;
}